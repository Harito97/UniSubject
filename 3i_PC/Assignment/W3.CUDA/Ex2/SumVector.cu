#include<stdio.h>

/* Main function, executed on host (CPU) */
int main(void)
{
    int size = 10;
    
    /* 1. allocate memory on GPU */

    float *d_A = NULL;
    if (hipMalloc((void **)&d_A, size) != hipSuccess)
        exit(EXIT_FAILURE);
    float *d_B = NULL;
    hipMalloc((void **)&d_B, size); 
    /* For clarity we'll not check for err */
    float *d_C = NULL;
    hipMalloc((void **)&d_C, size);


    /* 2. Copy data from Host to GPU */

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);


    /* 3. Execute GPU kernel */

    /* Calculate number of blocks and threads */
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) /
                        threadsPerBlock;
    /* Launch the Vector Add CUDA Kernel */
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B,
                                                  d_C, numElements);
    /* Wait for all the threads to complete */
    hipDeviceSynchronize();


    /* 4. Copy data from GPU back to Host */

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);


    /* 5. Free GPU memory */

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);


    return (0);
}