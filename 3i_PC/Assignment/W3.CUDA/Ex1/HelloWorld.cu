
#include <hip/hip_runtime.h>
#include <stdio.h>

/* Function executed on device (GPU */
__global__ void hello(void)
{
    printf("\tHello from GPU: thread %d and block %d\n",
           threadIdx.x,
           blockIdx.x);
}

/* Main function, executed on host (CPU) */
int main(void)
{
    int NUM_BLOCKS = 4;
    int BLOCK_WIDTH = 3;
    /* print message from CPU */
    printf("Hello Cuda!\n");
    /* execute function on device (GPU) */
    hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();
    /* wait until all threads finish their job */
    hipDeviceSynchronize();
    /* print message from CPU */
    printf("Welcome back to CPU!\n");
    return (0);
}
